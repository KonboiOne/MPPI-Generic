
#include <hip/hip_runtime.h>
//
// Created by jgibson37 on 2/7/20.
//

//#include "ar_robust_cost_kernel_test.cuh"

/*

__global__ void getCostmapCostTestKernel(ARRobustCost<>* cost, float* test_xu, float* cost_results, int num_points) {
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if(tid < num_points) {
    float* state = &test_xu[tid];
    float* control = &test_xu[tid+7];
    int crash = 0;
    float vars[2] = {1,1};
    float du[2] = {0,0};
    cost_results[tid] = cost.getCostmapCost(state, control, du, vars, &crash, tid);
  }
}

void launchGetCostmapCostTestKernel(ARRobustCost<>& cost, std::vector<std::array<float, 9>>& test_xu, std::vector<float>& cost_results) {
  int num_test_points = test_xu.size();
  cost_results.resize(num_test_points*9);

  float* cost_results_d;
  float* test_xu_d;
  HANDLE_ERROR(cudaMalloc((void**)&cost_results_d, sizeof(float)*num_test_points))
  HANDLE_ERROR(cudaMalloc((void**)&test_xu_d, sizeof(float)*9*num_test_points))

  for(int i = 0; i < num_test_points; i++) {
    for(int j = 0; j < 9; j++) {
      cost_results[9*i+j] = test_xu[i][j];
    }
  }

  HANDLE_ERROR(cudaMemcpy(test_xu_d, test_xu.data(), sizeof(float)*9*num_test_points, cudaMemcpyHostToDevice));

  // TODO amount should depend on the number of query points
  dim3 threadsPerBlock(num_test_points, 1);
  dim3 numBlocks(1, 1);
  getCostmapCostTestKernel<<<numBlocks,threadsPerBlock>>>(*cost.cost_d_, test_xu_d, cost_results_d, num_test_points);
  CudaCheckError();
  cudaDeviceSynchronize();

  // Copy the memory back to the host
  HANDLE_ERROR(cudaMemcpy(cost_results.data(), cost_results_d, sizeof(float)*num_test_points, cudaMemcpyDeviceToHost));

  cudaDeviceSynchronize();

  cudaFree(cost_results_d);
  cudaFree(test_xu_d);
}

*/